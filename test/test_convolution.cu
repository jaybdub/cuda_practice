#include <iostream>
#include "src/convolution.h"

using namespace std;

void Print(float * x, int x_len) {
  for (int i = 0; i < x_len; i++)
    cout << x[i] << endl;
}

void TestCorrelate1D() {
  const size_t N = 20;
  float a[N], b[N];
  for (int i = 0; i < N; i++)
    a[i] = i;
  const size_t N_w = 5;
  float w[N_w] = { -2, -1, 0, 1, 2 };
  float * d_a, * d_b, * d_w;   
  const size_t size = N * sizeof(float);
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_w, N_w * sizeof(float));
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
  hipMemcpy(d_w, w, N_w * sizeof(float), hipMemcpyHostToDevice);
  Correlate1D<<<N, 1>>>(d_a, N, d_w, N_w, d_b);
  hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
  Print(b, N);  
  hipFree(d_a);
  hipFree(d_b);
}

int main() {
  TestCorrelate1D();
  return 0;
}
